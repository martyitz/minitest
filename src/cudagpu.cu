#include "hip/hip_runtime.h"
#include "minitest.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime.h>
#include <>

int checkxfers();

__global__ void
xcompute(const double *d_l1, const double *d_r1, double *d_p1, int nelements )
{
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < nelements) {

#include "compute.h"

  }
}

void
twork( int iter, int threadnum)
{
  double *l1 = lptr[threadnum];
  double *r1 = rptr[threadnum];
  double *p1 = pptr[threadnum];

  hipError_t err = hipSuccess;

#if 0
  fprintf(stderr, "[%d]    T %d, I %d, start cuda twork\n",
      thispid, threadnum, iter );
#endif
  //allocate device memory for copying in l1 and r1, copying out p1
  size_t size = nn * sizeof(double);
  double *d_l1 = NULL;
  double *d_r1 = NULL;
  double *d_p1 = NULL;

  err = hipMalloc((void **)&d_l1, size);
  if (err != hipSuccess) {
    fprintf(stderr, "[%d]    T %d, I %d, Failed to allocate device vector d_l1 (error code %s)!\n",
      thispid, threadnum, iter, hipGetErrorString(err));
    exit(-1);
#if 0
  } else {
    fprintf(stderr, "[%d]    T %d, I %d, Alloc device vector d_l1 (%p)\n",
      thispid, threadnum, iter, d_l1 );
#endif
  }
  err = hipMalloc((void **)&d_r1, size);
  if (err != hipSuccess) {
    fprintf(stderr, "[%d]    T %d, I %d, Failed to allocate device vector d_r1 (error code %s)!\n",
      thispid, threadnum, iter, hipGetErrorString(err));
    exit(-1);
#if 0
  } else {
    fprintf(stderr, "[%d]    T %d, I %d, Alloc device vector d_r1 (%p)\n",
      thispid, threadnum, iter, d_r1 );
#endif
  }
  err = hipMalloc((void **)&d_p1, size);
  if (err != hipSuccess) {
    fprintf(stderr, "[%d]    T %d, I %d, Failed to allocate device vector d_p1 (error code %s)!\n",
      thispid, threadnum, iter, hipGetErrorString(err));
    exit(-1);
#if 0
  } else {
    fprintf(stderr, "[%d]    T %d, I %d, Alloc device vector d_p1 (%p)\n",
      thispid, threadnum, iter, d_p1 );
#endif
  }

  // Copy l1,r1 and p1 to the device
  err = hipMemcpy(d_l1, l1, size, hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    fprintf(stderr, "[%d]    T %d, I %d, Failed to copy device l1 to d_l1 (error code %s)!\n",
      thispid, threadnum, iter, hipGetErrorString(err));
    exit(-1);
#if 0
  } else {
    fprintf(stderr, "[%d]    T %d, I %d, Copied l1 to device vector d_l1\n",
      thispid, threadnum, iter );
#endif
  }
  err = hipMemcpy(d_r1, r1, size, hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    fprintf(stderr, "[%d]    T %d, I %d, Failed to copy device r1 to d_r1 (error code %s)!\n",
      thispid, threadnum, iter, hipGetErrorString(err));
    exit(-1);
#if 0
  } else {
    fprintf(stderr, "[%d]    T %d, I %d, Copied r1 to device vector d_r1\n",
      thispid, threadnum, iter );
#endif
  }
  err = hipMemcpy(d_p1, p1, size, hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    fprintf(stderr, "[%d]    T %d, I %d, Failed to copy device p1 to d_p1 (error code %s)!\n",
      thispid, threadnum, iter, hipGetErrorString(err));
    exit(-1);
#if 0
  } else {
    fprintf(stderr, "[%d]    T %d, I %d, Copied p1 to device vector d_p1\n",
      thispid, threadnum, iter );
#endif
  }

  // Set up and launch the CUDA kernel
  int threadsPerBlock = 256;
  int blocksPerGrid = ( nn + threadsPerBlock -1 ) / threadsPerBlock;

#if 0
  fprintf(stderr, "      [%d]  t %d, i%d, threadsPerBlock =  %d;  blocksPerGrid = %d\n",
    thispid, threadnum, iter, threadsPerBlock, blocksPerGrid );
#endif

  xcompute<<<blocksPerGrid, threadsPerBlock>>>(d_l1, d_r1, d_p1, nn);
  err = hipGetLastError();
  if (err != hipSuccess) {
    fprintf(stderr, "[%d]    T %d, I %d, Failed to launch compute kernel (error code %s)!\n",
      thispid, threadnum, iter, hipGetErrorString(err));
    exit(-1);
#if 0
  } else {
    fprintf(stderr, "[%d]    T %d, I %d, Ran GPU kernel for xcompute\n",
      thispid, threadnum, iter );
#endif
  }

  // Copy p1 back to the host
  err = hipMemcpy(p1, d_p1, size, hipMemcpyDeviceToHost);
  if (err != hipSuccess) {
    fprintf(stderr, "[%d]    T %d, I %d, Failed to copy p1 from device (error code %s)!\n",
      thispid, threadnum, iter, hipGetErrorString(err));
    exit(-1);
#if 0
  } else {
    fprintf(stderr, "[%d]    T %d, I %d, Copied d_p1 back to host\n",
      thispid, threadnum, iter );
#endif
  }

  // Free the device memory
  err = hipFree(d_l1);
  if (err != hipSuccess) {
    fprintf(stderr, "[%d]    T %d, I %d, Failed to free d_l1 from device (error code %s)!\n",
      thispid, threadnum, iter, hipGetErrorString(err));
    exit(-1);
#if 0
  } else {
    fprintf(stderr, "[%d]    T %d, I %d, Freed device vector d_l1 (%p)\n",
      thispid, threadnum, iter, d_l1 );
#endif
  }
  err = hipFree(d_r1);
  if (err != hipSuccess) {
    fprintf(stderr, "[%d]    T %d, I %d, Failed to free d_r1 from device (error code %s)!\n",
      thispid, threadnum, iter, hipGetErrorString(err));
    exit(-1);
#if 0
  } else {
    fprintf(stderr, "[%d]    T %d, I %d, Freed device vector d_r1 (%p)\n",
      thispid, threadnum, iter, d_r1 );
#endif
  }
  err = hipFree(d_p1);
  if (err != hipSuccess) {
    fprintf(stderr, "[%d]    T %d, I %d, Failed to free d_p1 from device (error code %s)!\n",
      thispid, threadnum, iter, hipGetErrorString(err));
    exit(-1);
#if 0
  } else {
    fprintf(stderr, "[%d]    T %d, I %d, Freed device vector d_p1 (%p)\n",
      thispid, threadnum, iter, d_p1 );
#endif
  }
#if 0
  fprintf(stderr, "    [%d] Completed iteration %d, thread %d\n\n",
    thispid, iter, threadnum);
#endif
  spacer (50, true);
}

void
initgpu()
{
#if 0
  /* determine number of GPU's */
  int numdev = omp_get_num_devices();
  fprintf (stderr,     "[%d] Machine has %d GPU device%s\n",
    thispid, numdev, (numdev==1 ? "" : "s") );

  /* Test if GPU is available */
  int	idev = omp_is_initial_device();

  int runningOnGPU = -1;
  #pragma omp target map(from:runningOnGPU)
  {
    runningOnGPU = omp_is_initial_device();
  }

  /* If still running on CPU, GPU must not be available */
  if (runningOnGPU != 0) {
#ifndef IGNORE_BAD_INITIAL_DEVICE
    fprintf(stderr, "[%d] ERROR unable to use the GPU! idev = %d, runningOnGpU -- omp_is_initial_device() = %d; exiting\n",
      thispid, idev, runningOnGPU);
    exit(1);
#else
    fprintf(stderr, "[%d] ignoring error unable to use gpu! idev = %d, runningOnGpU -- omp_is_initial_device() = %d; trying anyway\n",
      thispid, idev, runningOnGpu );
#endif
  } else {
    fprintfstderr, "   [%d] gputest is able to use the GPU! idev = %d, runningOnGpU -- omp_is_initial_device()\n",
      thispid, idev );
  }

  int ret = checkxfers();
  if (ret != 0 ) {
    fprintf(stderr, "[%d] Return from checkxfers = %d\n", thispid,, ret);
  }
#endif
}

int
checkxfers()
{
// This is only needed for the OpenMP offload version
  return 0;
}
