#include "hip/hip_runtime.h"
#include "minitest.h"
#include <hip_runtime_api.h>
#include <hip_runtime.h>
#include <hip/device_functions.h>

int checkxfers();

__global__ void
xcompute(const double *d_l1, const double *d_r1, double *d_p1, int nelements )
{
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < nelements) {

#include "compute.h"

  }
}

void
twork( int iter, int threadnum)
{
  double *l1 = lptr[threadnum];
  double *r1 = rptr[threadnum];
  double *p1 = pptr[threadnum];

  hipError_t err = hipSuccess;

  hrtime_t starttime = gethrtime();

  //allocate device memory for copying in l1 and r1, copying out p1
  size_t size = nn * sizeof(double);
  double *d_l1 = NULL;
  double *d_r1 = NULL;
  double *d_p1 = NULL;
  err = hipMalloc((void **)&d_l1, size);
  if (err != hipSuccess) {
    fprintf(stderr, "[%d] Failed to allocate device vector d_l1 (error code %s)!\n", thispid, hipGetErrorString(err));
    exit(-1);
#if 0
  } else {
    fprintf(stderr, "[%d] Allocated device vector d_l1\n", thispid );
#endif
  }
  err = hipMalloc((void **)&d_r1, size);
  if (err != hipSuccess) {
    fprintf(stderr, "[%d] Failed to allocate device vector d_r1 (error code %s)!\n", thispid, hipGetErrorString(err));
    exit(-1);
#if 0
  } else {
    fprintf(stderr, "[%d] Allocated device vector d_r1\n", thispid );
#endif
  }
  err = hipMalloc((void **)&d_p1, size);
  if (err != hipSuccess) {
    fprintf(stderr, "[%d] Failed to allocate device vector d_p1 (error code %s)!\n", thispid, hipGetErrorString(err));
    exit(-1);
#if 0
  } else {
    fprintf(stderr, "[%d] Allocated device vector d_p1\n", thispid );
#endif
  }

  // Copy l1,r1 and p1 to the device
  err = hipMemcpy(d_l1, l1, size, hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    fprintf(stderr, "[%d] Failed to copy device l1 to d_l1 (error code %s)!\n", thispid, hipGetErrorString(err));
    exit(-1);
#if 0
  } else {
    fprintf(stderr, "[%d] Copied l1 to device vector d_l1\n", thispid );
#endif
  }
  err = hipMemcpy(d_r1, r1, size, hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    fprintf(stderr, "[%d] Failed to copy device r1 to d_r1 (error code %s)!\n", thispid, hipGetErrorString(err));
    exit(-1);
#if 0
  } else {
    fprintf(stderr, "[%d] Copied r1 to device vector d_r1\n", thispid );
#endif
  }
  err = hipMemcpy(d_p1, p1, size, hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    fprintf(stderr, "[%d] Failed to copy device p1 to d_p1 (error code %s)!\n", thispid, hipGetErrorString(err));
    exit(-1);
#if 0
  } else {
    fprintf(stderr, "[%d] Copied p1 to device vector d_p1\n", thispid );
#endif
  }

  // Set up and launch the HIP kernel
  int threadsPerBlock = 256;
  int blocksPerGrid = ( nn + threadsPerBlock -1 ) / threadsPerBlock;

#if 0
  fprintf(stderr, "      [%d] threadsPerBlock =  %d;  blocksPerGrid = %d\n", thispid, threadsPerBlock, blocksPerGrid );
#endif

  xcompute<<<blocksPerGrid, threadsPerBlock>>>(d_l1, d_r1, d_p1, nn);
  err = hipGetLastError();
  if (err != hipSuccess) {
    fprintf(stderr, "[%d] Failed to launch compute kernel (error code %s)!\n", thispid, hipGetErrorString(err));
    exit(-1);
#if 0
  } else {
    fprintf(stderr, "[%d] Ran GPU kernel for xcompute\n", thispid );
#endif
  }

  // Copy p1 back to the host
  err = hipMemcpy(p1, d_p1, size, hipMemcpyDeviceToHost);
  if (err != hipSuccess) {
    fprintf(stderr, "[%d] Failed to copy p1 from device (error code %s)!\n", thispid, hipGetErrorString(err));
    exit(-1);
#if 0
  } else {
    fprintf(stderr, "[%d] Copied d_p1 back to host\n", thispid );
#endif
  }

  // Free the device memory
  err = hipFree(d_l1);
  if (err != hipSuccess) {
    fprintf(stderr, "[%d] Failed to free d_l1 from device (error code %s)!\n", thispid, hipGetErrorString(err));
    exit(-1);
#if 0
  } else {
    fprintf(stderr, "[%d] Freed device vector d_l1\n", thispid );
#endif
  }
  err = hipFree(d_r1);
  if (err != hipSuccess) {
    fprintf(stderr, "[%d] Failed to free d_r1 from device (error code %s)!\n", thispid, hipGetErrorString(err));
    exit(-1);
#if 0
  } else {
    fprintf(stderr, "[%d] Freed device vector d_r1\n", thispid );
#endif
  }
  err = hipFree(d_p1);
  if (err != hipSuccess) {
    fprintf(stderr, "[%d] Failed to free d_p1 from device (error code %s)!\n", thispid, hipGetErrorString(err));
    exit(-1);
#if 0
  } else {
    fprintf(stderr, "[%d] Freed device vector d_p1\n", thispid );
#endif
  }

  hrtime_t endtime = gethrtime();
  double  tempus =  (double) (endtime - starttime) / (double)1000000000.;
#if 1
  fprintf(stderr, "    [%d] Completed iteration %d, thread %d in %13.9f s.\n\n",
    thispid, iter, threadnum, tempus);
#endif
  spacer(50, true);
}

void
initgpu()
{
#if 0
  /* determine number of GPU's */
  int numdev = omp_get_num_devices();
  fprintf (stderr, "    [%d] Machine has %d GPU device%s\n", thispid, numdev, (numdev==1 ? "" : "s") );

  /* Test if GPU is available */
  int	idev = omp_is_initial_device();

  int runningOnGPU = -1;
  #pragma omp target map(from:runningOnGPU)
  {
    runningOnGPU = omp_is_initial_device();
  }

  /* If still running on CPU, GPU must not be available */
  if (runningOnGPU != 0) {
#ifndef IGNORE_BAD_INITIAL_DEVICE
    fprintf(stderr, "[%d] ERROR unable to use the GPU! idev = %d, runningOnGpU -- omp_is_initial_device() = %d; exiting\n",
      thispid, idev, runningOnGPU);
    exit(1);
#else
    fprintf(stderr, "[%d] ignore error unable to use gpu! idev = %d, runningOnGpU -- omp_is_initial_device() = %d; trying anyway\n",
      thispid, idev, runningOnGPU);
#endif
  } else {
    fprintfstderr, "    [%d] gputest is able to use the GPU! idev = %d, runningOnGpU -- omp_is_initial_device()\n", thispid, idev );
  }

  int ret = checkxfers();
  if (ret != 0 ) {
    fprintf(stderr, "[%d] Return from checkxfers = %d\n", thispid, ret);
  }
#endif
}

int
checkxfers()
{
// This is only needed for the OpenMP version
  return 0;
}
